#include "hip/hip_runtime.h"
#include <cub/block/block_reduce.cuh>
#include <iostream>

#define checkCUDA(expression)                               \
  {                                                         \
    hipError_t status = (expression);                      \
    if (status != hipSuccess) {                            \
      std::cerr << "Error on line " << __LINE__ << ": "     \
                << hipGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                              \
    }                                                       \
  }

template <typename T>
void IsClose2DHost(const T* x, const T* y, int N, int C, int D, std::string msg,
                   float atol, float rtol);

template <typename T>
void Print2DHost(const T* x, int N, int C, int D, std::string msg);

template <typename T, typename U>
void InstanceNormCPU(const T* x, const U* gamma, const U* beta, const int N, const int C,
                    const int D, const U epsilon, T* y);

template <typename T, typename U>
void InstanceNormGradCPU(const T* dy, const T* x, const U* gamma, const int N, const int C,
                      const int D, const U epsilon, U* dgamma, U* dbeta, T* dx);

template <typename T, typename U>
void InstanceNormCPUHelper(const T* x, const U* gamma, const U* beta, const int N, const int C,
                        const int D, const U epsilon, T* y_h) {
  T* x_h = new T[N * C * D];
  U* gamma_h = new U[C];
  U* beta_h = new U[C];

  checkCUDA(hipMemcpy(x_h, x, N * C * D * sizeof(T), hipMemcpyDeviceToHost));
  checkCUDA(hipMemcpy(gamma_h, gamma, C * sizeof(U), hipMemcpyDeviceToHost));
  checkCUDA(hipMemcpy(beta_h, beta, C * sizeof(U), hipMemcpyDeviceToHost));

  InstanceNormCPU(x_h, gamma_h, beta_h, N, C, D, epsilon, y_h);

  delete[] x_h;
  delete[] gamma_h;
  delete[] beta_h;
}

template <typename T, typename U>
void InstanceNormGradCPUHelper(const T* dy, const T* x, const U* gamma,
                            const int N, const int C, const int D, const U epsilon,
                            U* dgamma_h, U* dbeta_h, T* dx_h) {
  T* dy_h = new T[N * C * D];
  T* x_h = new T[N * C * D];
  U* gamma_h = new U[C];
  checkCUDA(hipMemcpy(dy_h, dy, N * C * D * sizeof(T), hipMemcpyDeviceToHost));
  checkCUDA(hipMemcpy(x_h, x, N * C * D * sizeof(T), hipMemcpyDeviceToHost));
  checkCUDA(hipMemcpy(gamma_h, gamma, C * sizeof(U), hipMemcpyDeviceToHost));

  InstanceNormGradCPU(dy_h, x_h, gamma_h, N, C, D, epsilon, dgamma_h, dbeta_h, dx_h);

  delete[] dy_h;
  delete[] x_h;
  delete[] gamma_h;
}

const int kBlockSize = 128;
const int kWarpSize = 32;

int DivUp(int a, int b) { return (a + b - 1) / b; }

template <typename T>
void PrepareAlloc(T** x, int size, int init = -1) {
  srand(12);
  T* buf = new T[size];
  for (int i = 0; i < size; i++) {
    if (init != -1) {
      buf[i] = init;
    } else {
      buf[i] = static_cast<T>(static_cast<float>(rand()) / RAND_MAX);
    }
  }

  checkCUDA(hipMalloc(&(*x), size * sizeof(T)));
  checkCUDA(hipMemcpy(*x, buf, size * sizeof(T), hipMemcpyHostToDevice));

  delete[] buf;
}

template <typename T>
void Print2D(const T* x, int N, int C, int D, std::string msg) {
  T* buf = new T[N * D];
  checkCUDA(hipMemcpy(buf, x, N * C * D * sizeof(T), hipMemcpyDeviceToHost));
  Print2DHost(buf, N, C, D, msg);
  delete[] buf;
}

template <typename T>
void IsClose2D(const T* x, const T* y, int N, int C, int D, std::string msg,
               float atol = 1e-3, float rtol = 1e-3) {
  if (D == 10000000) {  // Mainly for y when NxD=10x10000000
    atol = 1e-1;
  }
  if (D == 1000000) {  // Mainly for y when NxD=100x1000000
    atol = 1e-2;
  }
  if (D == 10) {  // Mainly for dgamma when NxD=10000000x10
    atol = 1e-1;
  }
  T* buf = new T[N * C * D];
  checkCUDA(hipMemcpy(buf, x, N * C * D * sizeof(T), hipMemcpyDeviceToHost));
  IsClose2DHost(buf, y, N, C, D, msg, atol, rtol);
  delete[] buf;
}

template <typename T, typename U>
__host__ __device__ U GetAs(const T* __restrict__ in, int offset) {
  return static_cast<U>(in[offset]);
}
///////////////////////////// MAYBE inspect
template <typename T, typename U>
struct MeanOp {
  int D;
  __device__ U Compute(const T* x, const int& row, const int& col) const {
    return GetAs<T, U>(x, row * D + col);
  }
  __device__ U Finalize(const U& sum) const { return sum / D; }
};

template <typename T, typename U>
struct IvarOp {
  const U* cache_mean;
  int D;
  U epsilon;
  __device__ U Compute(const T* x, const int& row, const int& col,
                       const U& mean) const {
    U curr = GetAs<T, U>(x, row * D + col);
    return (curr - mean) * (curr - mean);
  }
  __device__ U Compute(const T* x, const int& row, const int& col) const {
    return Compute(x, row, col, cache_mean[row]);
  }
  __device__ U Finalize(const U& sum) const { return rsqrt(sum / D + epsilon); }
};

template <typename T, typename U>
struct DvarOp {
  const U* gamma;
  const T* x;
  const U* cache_ivar;
  const U* cache_mean;
  int D;
  __device__ U Compute(const T* dy, const int& row, const int& col) const {
    U curr = GetAs<T, U>(dy, row * D + col);
    return curr * gamma[col] * (x[row * D + col] - cache_mean[row]) * (-0.5) *
           (cache_ivar[row] * cache_ivar[row] * cache_ivar[row]);
  }
  __device__ U Finalize(const U& sum) const { return sum; }
};

template <typename T, typename U>
struct DmeanOp {
  const U* gamma;
  const T* x;
  const U* cache_ivar;
  const U* cache_mean;
  const U* dl_dvars;
  int D;
  __device__ U Compute(const T* dy, const int& row, const int& col,
                       const U& dl_dvar) const {
    U curr = GetAs<T, U>(dy, row * D + col);
    return -1. * curr * gamma[col] * cache_ivar[row] +
           dl_dvar * (-2. / D) * (x[row * D + col] - cache_mean[row]);
  }
  __device__ U Compute(const T* dy, const int& row, const int& col) const {
    return Compute(dy, row, col, dl_dvars[row]);
  }
  __device__ U Finalize(const U& sum) const { return sum; }
};

template <typename T, typename U>
struct DxOp {
  const T* x;
  const U* cache_mean;
  const U* cache_ivar;
  const U* gamma;
  const U* dl_dvars;
  const U* dl_dmus;
  int D;
  __device__ T Compute(const T* dy, const int& row, const int& col) const {
    U curr = GetAs<T, U>(dy, row * D + col);
    U dl_di = curr * gamma[col] * cache_ivar[row];
    U di_dx = 1.;
    U dvar_dx = 2. * (x[row * D + col] - cache_mean[row]) / D;
    U dmu_dx = 1. / D;
    U dl_dx = dl_di * di_dx + dl_dvars[row] * dvar_dx + dl_dmus[row] * dmu_dx;
    return static_cast<T>(dl_dx);
  }
};

template <typename T, typename U>
struct YOp {
  const U* cache_mean;
  const U* cache_ivar;
  const U* gamma;
  const U* beta;
  int D;
  __device__ T Compute(const T* x, const int& row, const int& col) const {
    U mean = cache_mean[row];
    U ivar = cache_ivar[row];
    U curr = GetAs<T, U>(x, row * D + col);
    return static_cast<T>((curr - mean) * ivar * gamma[col] + beta[col]);
  }
};

template <typename T, typename Op>
__global__ void InstanceNormUpdate(const T* __restrict__ in, const int N, const int C,
                                const int D, T* out, Op op) {
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;

  if (tid >= N * C * D) return;

  const int col = tid % D;
  const int row = tid / D;
  out[tid] = op.Compute(in, row, col);
}

template <typename T, typename U>
void InstanceNormGPU(const T* x, const U* gamma, const U* beta, const U epsilon,
                  const int N, const int C, const int D, T* y, U* cache_mean,
                  U* cache_ivar) {
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  bool use_single_warp = (D <= kWarpSize);

  const int min_num_blocks = kWarpSize;
  const int min_workload_per_thread = 100;
  bool use_single_block =
      (D <= min_num_blocks * kBlockSize * min_workload_per_thread);

  MeanOp<T, U> mean_ops{D};
  IvarOp<T, U> ivar_ops{cache_mean, D, epsilon};

  hipEventRecord(start);
  int NxC = N * C;
  if (use_single_warp) {
    printf("XLOG: Mean/Var -> single-warp per row\n");
    InstanceNormRowReduceInToOutWarp<<<DivUp(NxC, kBlockSize / kWarpSize),
                                    kBlockSize>>>(
        x, N, C, D, cache_mean, cache_ivar, mean_ops, ivar_ops);
  } else if (use_single_block) {
    // printf("XLOG: Mean/Var -> single-block per row\n");
    // LayerNormRowReduceInToOut<<<N, kBlockSize>>>(
    //     x, N, D, cache_mean, cache_ivar, mean_ops, ivar_ops);
  } else {
    // printf("XLOG: Mean/Var -> multi-block per row\n");
    // const int blocks_per_row = DivUp(D, kBlockSize * min_workload_per_thread);

    // float* temp_sum;
    // float* temp_ivar;
    // PrepareAlloc(&temp_sum, N * blocks_per_row);
    // PrepareAlloc(&temp_ivar, N * blocks_per_row);

    // dim3 threads(kBlockSize, 1, 1);
    // dim3 blocks(blocks_per_row, N, 1);
    // printf("XLOG: num_blocks per row=%d\n", blocks.x);

    // // For long rows, we launch n blocks to process each row. The intermediate
    // // results are stored in a temp memory with the size of N*n. Then, we launch
    // // single block to handle each row of the temp memory.
    // LayerNormRowReduceInToTemp<<<blocks, threads>>>(x, N, D, temp_sum,
    //                                                 mean_ops);
    // LayerNormRowReduceTempToOut<<<N, threads>>>(temp_sum, N, blocks_per_row,
    //                                             cache_mean, mean_ops);

    // LayerNormRowReduceInToTemp<<<blocks, threads>>>(x, N, D, temp_ivar,
    //                                                 ivar_ops);
    // LayerNormRowReduceTempToOut<<<N, threads>>>(temp_ivar, N, blocks_per_row,
    //                                             cache_ivar, ivar_ops);

    // checkCUDA(hipFree(temp_ivar));
    // checkCUDA(hipFree(temp_sum));
  }
  hipEventRecord(stop);

  hipEventSynchronize(stop);
  float milliseconds_reduce = 0;
  hipEventElapsedTime(&milliseconds_reduce, start, stop);

  hipEventRecord(start);
  YOp<T, U> y_ops{cache_mean, cache_ivar, gamma, beta, D};
  InstanceNormUpdate<<<DivUp(N * D, kBlockSize), kBlockSize>>>(x, N, C, D, y, y_ops);
  hipEventRecord(stop);

  hipEventSynchronize(stop);
  float milliseconds_update = 0;
  hipEventElapsedTime(&milliseconds_update, start, stop);
  printf("InstanceNormGPU time %.2f ms (reduce=%f, update=%f)\n",
         milliseconds_reduce + milliseconds_update, milliseconds_reduce,
         milliseconds_update);
}

template <typename T, typename U, typename Op1, typename Op2>
__global__ void InstanceNormRowReduceInToOutWarp(const T* __restrict__ in,
                                              const int N, const int C,
                                              const int D, U* out1,
                                              U* out2, Op1 op1, Op2 op2) {
  // cache_mean, cache_ivar, mean_ops, ivar_ops
  const int tid = threadIdx.x % kWarpSize;

  const int num_warps = kBlockSize / kWarpSize;
  typedef hipcub::WarpReduce<U> WarpReduce;
  typename WarpReduce::TempStorage temp_storage[num_warps];

  const int local_warp_id = threadIdx.x / kWarpSize;
  const int warp_id = blockIdx.x * num_warps + local_warp_id;
  int NxC = N * C;
  for (int k = warp_id; k < NxC; k += gridDim.x * num_warps) {
    U partial_sum = 0;
    for (int i = tid; i < D; i += kWarpSize) {
      partial_sum += op1.Compute(in, k, i);
    }

    U sum = WarpReduce(temp_storage[local_warp_id]).Sum(partial_sum);

    sum = hipcub::ShuffleIndex<kWarpSize>(sum, 0, 0xffffffff);
    sum = op1.Finalize(sum);
    if (tid == 0) {
      out1[k] = sum;
    }

    partial_sum = 0;
    for (int i = tid; i < D; i += kWarpSize) {
      partial_sum += op2.Compute(in, k, i, sum);
    }

    sum = WarpReduce(temp_storage[local_warp_id]).Sum(partial_sum);

    if (tid == 0) {
      out2[k] = op2.Finalize(sum);
    }
  }
}

#define DTYPE float

int main(int argc, char** argv) {
  /** Parameters and Knobs **/
  int N = 2;
  int C = 3;
  int D = 4;
  if (argc >= 3) {
    N = atoi(argv[1]);
    C = atoi(argv[2]);
    D = atoi(argv[3]);
  }
  bool allow_print = false;

  DTYPE* x;
  float* gamma;
  float* beta;
  PrepareAlloc(&x, N * C * D);
  PrepareAlloc(&gamma, C);
  PrepareAlloc(&beta, C);

  DTYPE* y;
  float* cache_ivar;
  float* cache_mean;
  PrepareAlloc(&y, N * C * D);
  PrepareAlloc(&cache_ivar, N * C);
  PrepareAlloc(&cache_mean, N * C);

  const float epsilon = 0.001f;
  InstanceNormGPU(x, gamma, beta, epsilon, N, C, D, y, cache_mean, cache_ivar);
  if (allow_print) {
    Print2D(y, N, C, D, "GPU y:");
  }

  DTYPE* y_h = new DTYPE[N * D];
  InstanceNormCPUHelper(x, gamma, beta, N, C, D, epsilon, y_h);
  if (allow_print) {
    Print2DHost(y_h, N, C, D, "CPU y:");
  }
  IsClose2D(y, y_h, N, C, D, "y");
  delete[] y_h;
  // ---- Forward Done Here ----

  // DTYPE* dy;
  // PrepareAlloc(&dy, N * D, 1);

  // DTYPE* dx;
  // float* dgamma;
  // float* dbeta;
  // PrepareAlloc(&dx, N * D);
  // PrepareAlloc(&dgamma, D);
  // PrepareAlloc(&dbeta, D);

  // LayerNormGradGPU(dy, x, cache_mean, cache_ivar, gamma, N, D, dx, dgamma,
  //                  dbeta);
  // if (allow_print) {
  //   Print2D(dgamma, 1, D, "GPU dgamma:");
  //   Print2D(dbeta, 1, D, "GPU dbeta:");
  //   Print2D(dx, N, D, "GPU dx:");
  // }

  // DTYPE* dx_h = new DTYPE[N * D];
  // float* dgamma_h = new float[D];
  // float* dbeta_h = new float[D];
  // LayerNormGradCPUHelper(dy, x, gamma, N, D, epsilon, dgamma_h, dbeta_h, dx_h);
  // if (allow_print) {
  //   Print2DHost(dgamma_h, 1, D, "CPU dgamma:");
  //   Print2DHost(dbeta_h, 1, D, "CPU dbeta:");
  //   Print2DHost(dx_h, N, D, "CPU dx:");
  // }

  // IsClose2D(dgamma, dgamma_h, 1, D, "dgamma");
  // IsClose2D(dbeta, dbeta_h, 1, D, "dbeta");
  // IsClose2D(dx, dx_h, N, D, "dx");

  // delete[] dx_h;
  // delete[] dgamma_h;
  // delete[] dbeta_h;
  // // ---- Backward Done Here ----

  checkCUDA(hipFree(x));
  checkCUDA(hipFree(gamma));
  checkCUDA(hipFree(beta));
  checkCUDA(hipFree(y));
  // checkCUDA(hipFree(dy));
  // checkCUDA(hipFree(dx));
  // checkCUDA(hipFree(dgamma));
  // checkCUDA(hipFree(dbeta));
  checkCUDA(hipFree(cache_mean));
  checkCUDA(hipFree(cache_ivar));
}
